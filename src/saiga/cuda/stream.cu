/**
 * Copyright (c) 2017 Darius Rückert
 * Licensed under the MIT License.
 * See LICENSE file for more information.
 */

#include "stream.h"

#ifndef _WIN32
#include <nvToolsExtCudaRt.h>
#endif




namespace Saiga
{
namespace CUDA
{
CudaStream::CudaStream()
{
    hipStreamCreate(&stream);
}

CudaStream::~CudaStream()
{
    hipStreamDestroy(stream);
}

void CudaStream::waitForEvent(hipEvent_t event)
{
    hipStreamWaitEvent(stream, event, 0);
}

void CudaStream::synchronize()
{
    hipStreamSynchronize(stream);
}


hipStream_t CudaStream::legacyStream()
{
    return hipStreamLegacy;
}

hipStream_t CudaStream::perThreadStream()
{
    return hipStreamPerThread;
}

Saiga::CUDA::CudaStream::operator hipStream_t() const
{
    return stream;
}

void CudaStream::setName(const std::string& name) { 
#ifndef _WIN32
	nvtxNameCudaStreamA(stream, name.c_str());
#endif
}

}  // namespace CUDA
}  // namespace Saiga
